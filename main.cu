#include <stdio.h>
#include "CudaUtils.cu.h"
#include "functors.cu.h"

void testything()
{
	int a[]={13,17,16,19,12, 21, 14, 15, 20, 22};
	int n=10;
	int b[10];
	int * ad, * bd, * outlen;

	hipMalloc((void**) &ad, n*sizeof(int));
	hipMalloc((void**) &bd, n*sizeof(int));
	hipMalloc((void**) &outlen, sizeof(int));
	
	hipMemcpy(ad,a,n*sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(bd,b,n*sizeof(int),hipMemcpyHostToDevice);

	cudaMike::filter<<<1,n,256>>>(ad,n,bd,outlen, cudaMike::in_range<int>(16,21));

	int out;	
	hipMemcpy(&out,outlen,sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(b,bd,out*sizeof(int),hipMemcpyDeviceToHost);
	
	for(int i=0; i<out; ++i)
		printf("%d:%d \t", i, b[i]);
	printf("\n");
	hipFree(a);
	hipFree(b);
	hipFree(outlen);
}
